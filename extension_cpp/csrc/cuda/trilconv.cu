#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace extension_cpp {


__global__ void trilconv_kernel(int numel, const float* input, const float* conv_weights, float* result){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < numel) {
    for (int i=0; i<idx; i++){
      result[idx] += input[i] * conv_weights[i];
    }
  }
}

at::Tensor trilconv_kernel(const at::Tensor& a, const at::Tensor& b) {
  TORCH_CHECK(a.sizes() == b.sizes());
  TORCH_CHECK(a.dtype() == at::kFloat);
  TORCH_CHECK(b.dtype() == at::kFloat);
  TORCH_INTERNAL_ASSERT(a.device().type() == at::DeviceType::CUDA);
  TORCH_INTERNAL_ASSERT(b.device().type() == at::DeviceType::CUDA);
  at::Tensor a_contig = a.contiguous();
  at::Tensor b_contig = b.contiguous();
  at::Tensor result = torch::empty(a_contig.sizes(), a_contig.options());
  const float* a_ptr = a_contig.data_ptr<float>();
  const float* b_ptr = b_contig.data_ptr<float>();
  float* result_ptr = result.data_ptr<float>();
  int numel = a_contig.numel();
  mul_kernel<<<(numel+255)/256, 256>>>(numel, a_ptr, b_ptr, result_ptr);
  return result;
}

// Registers CUDA implementations for mymuladd, mymul, myadd_out
TORCH_LIBRARY_IMPL(extension_cpp, CUDA, m) {
  m.impl("trilconv", &trilconv_kernel);
}
}
